#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define MAX 4
#define DIMENSION 10000

 void initializeMatrix(int* matrix) {
    srand(time(0));
	for (int i = 0; i < DIMENSION; i++)
	{
		for (int j = 0; j < DIMENSION; j++) 
		{
			matrix[i * DIMENSION + j] = rand() % MAX;
		}
	}
}
void clearMatrix(int* matrix) {
	for (int i = 0; i < DIMENSION; i++)
	{
		for (int j = 0; j < DIMENSION; j++) 
		{
			matrix[i * DIMENSION + j] = 0;
		}
	}
}

__global__ void matrix_mul_kernel(int* a, int* b, int* c)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	
	if (row < DIMENSION && col < DIMENSION)
	{
		int result = 0;
		for (int k = 0; k < DIMENSION; k++)
		{
			result += (a[row * DIMENSION + k] * b[k * DIMENSION + col]);
		}
		c[row * DIMENSION + col] = result;
	}
}

int main(int argc, char **argv)
{
    int size = DIMENSION * DIMENSION;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

	int* hostA;
	int* hostB;
	int* hostC;
	int* deviceA;
	int* deviceB;
	int* deviceC;
	
	hostA = (int*)malloc(size * sizeof(int));
	hostB = (int*)malloc(size * sizeof(int));
	hostC = (int*)malloc(size * sizeof(int));
	
	int i, j;
	
	initializeMatrix(hostA);
	initializeMatrix(hostB);
    clearMatrix(hostC);

	hipMalloc((void **)&deviceA, size * sizeof(int));
	hipMalloc((void **)&deviceB, size * sizeof(int));
	hipMalloc((void **)&deviceC, size * sizeof(int));

	hipMemcpy(deviceA, hostA, size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(deviceB, hostB, size * sizeof(int), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(2, 2);
    dim3 blocksPerGrid((int)ceil(DIMENSION/2),(int)ceil(DIMENSION/2));

    hipEventRecord(start);
    matrix_mul_kernel<<<blocksPerGrid,threadsPerBlock>>>(deviceA, deviceB, deviceC);
    hipEventRecord(stop);

	hipMemcpy(hostC, deviceC, size * sizeof(int), hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
	
	hipFree(deviceA);
	hipFree(deviceB);
	hipFree(deviceC);

	// //print matrix A
    // printf("Matrix A: \n");
	// for (i = 0; i < DIMENSION; i++)
	// {
	// 	for (j = 0; j < DIMENSION; j++)
	// 	{
	// 		printf("%d ", hostA[i * DIMENSION + j]);
	// 	}
	// 	printf("\n");
	// }
    // printf("\n");
    // //print matrix B
    // printf("Matrix B: \n");
	// for (i = 0; i < DIMENSION; i++)
	// {
	// 	for (j = 0; j < DIMENSION; j++)
	// 	{
	// 		printf("%d ", hostB[i * DIMENSION + j]);
	// 	}
	// 	printf("\n");
	// }
    // printf("\n");
    // //print the resulting matrix
    // printf("Matrix C: \n");
	// for (i = 0; i < DIMENSION; i++)
	// {
	// 	for (j = 0; j < DIMENSION; j++)
	// 	{
	// 		printf("%d ", hostC[i * DIMENSION + j]);
	// 	}
	// 	printf("\n");
	// }
    // printf("\n");
    printf("Time [ms]: %f \n", milliseconds);
}

