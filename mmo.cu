#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define MAX 4
#define DIMENSION 10000

#define TILE_SIZE 4

 void initializeMatrix( int* matrix) {
    srand(time(0));
	for (int i = 0; i < DIMENSION; i++)
	{
		for (int j = 0; j < DIMENSION; j++) 
		{
			matrix[i * DIMENSION + j] = rand() % MAX;
		}
	}
}

void clearMatrix( int* matrix) {
	for (int i = 0; i < DIMENSION; i++)
	{
		for (int j = 0; j < DIMENSION; j++) 
		{
			matrix[i * DIMENSION + j] = 0;
		}
	}
}

__global__ void matrix_mul_kernel(int* a, int* b, int* c)
{
	__shared__ int sharedA[TILE_SIZE][TILE_SIZE];
	__shared__ int sharedB[TILE_SIZE][TILE_SIZE];

	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	
	if (row < DIMENSION && col < DIMENSION)
	{
		int result = 0;
		int k;
		int phase;
		
		for (phase = 0; phase <= DIMENSION/TILE_SIZE; phase++)
		{
			sharedA[ty][tx] = a[row * DIMENSION + phase * TILE_SIZE + tx];
			sharedB[ty][tx] = b[(phase * TILE_SIZE + ty) * DIMENSION + col];

			__syncthreads();
			for (k = 0; k < TILE_SIZE; k++)
			{
				if (k + (phase * TILE_SIZE) < DIMENSION) 
				{
					result += (sharedA[ty][k] * sharedB[k][tx]);
				}
			}
			__syncthreads();
		}	
		c[row * DIMENSION + col] = result;
	}
}


int main(int argc, char **argv)
{
    int size = DIMENSION * DIMENSION;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

	int* hostA;
	int* hostB;
	int* hostC;
	int* deviceA;
	int* deviceB;
	int* deviceC;

	hostA = (int*)malloc(size * sizeof(int));
	hostB = (int*)malloc(size * sizeof(int));
	hostC = (int*)malloc(size * sizeof(int));

	int i, j;
	
	initializeMatrix(hostA);
	initializeMatrix(hostB);
    clearMatrix(hostC);

	hipMalloc((void **)&deviceA, size * sizeof(int));
	hipMalloc((void **)&deviceB, size * sizeof(int));
	hipMalloc((void **)&deviceC, size * sizeof(int));

	hipMemcpy(deviceA, hostA, size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(deviceB, hostB, size * sizeof(int), hipMemcpyHostToDevice);

	int tpg;
	int bpg;

	if(DIMENSION < 100)
	{
		tpg = DIMENSION;
		bpg = 1;
	}
	else
	{
		tpg = TILE_SIZE;
		bpg = (int)ceil(DIMENSION/TILE_SIZE);
	}

	dim3 threadsPerBlock(tpg, tpg);
	dim3 blocksPerGrid(bpg, bpg);

    hipEventRecord(start);
    matrix_mul_kernel<<<blocksPerGrid, threadsPerBlock>>>(deviceA, deviceB, deviceC);
    hipEventRecord(stop);

	hipMemcpy(hostC, deviceC, size * sizeof(int), hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
	
	hipFree(deviceA);
	hipFree(deviceB);
	hipFree(deviceC);

	// //print matrix A
    // printf("Matrix A: \n");
	// for (i = 0; i < DIMENSION; i++)
	// {
	// 	for (j = 0; j < DIMENSION; j++)
	// 	{
	// 		printf("%d ", hostA[i * DIMENSION + j]);
	// 	}
	// 	printf("\n");
	// }
    // printf("\n");
    // //print matrix B
    // printf("Matrix B: \n");
	// for (i = 0; i < DIMENSION; i++)
	// {
	// 	for (j = 0; j < DIMENSION; j++)
	// 	{
	// 		printf("%d ", hostB[i * DIMENSION + j]);
	// 	}
	// 	printf("\n");
	// }
    // printf("\n");
    //print the resulting matrix
    // printf("Matrix C: \n");
	// for (i = 0; i < DIMENSION; i++)
	// {
	// 	for (j = 0; j < DIMENSION; j++)
	// 	{
	// 		printf("%d ", hostC[i * DIMENSION + j]);
	// 	}
	// 	printf("\n");
	// }
    // printf("\n");
    printf("Time [ms]: %f \n", milliseconds);
}

