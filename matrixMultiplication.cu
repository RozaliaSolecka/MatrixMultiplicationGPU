#include "hip/hip_runtime.h"
/*
CUDA - generate array of random numbers and calculate occurence of odd and even numbers - no streams
*/
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define MAX 4
#define DIMENSION 1000

 void initializeMatrix( int* matrix) {
    srand(time(0));
	for (int i = 0; i < DIMENSION; i++)
	{
		for (int j = 0; j < DIMENSION; j++) 
		{
			matrix[i * DIMENSION + j] = rand() % MAX;
		}
	}
}
void clearMatrix( int* matrix) {
	for (int i = 0; i < DIMENSION; i++)
	{
		for (int j = 0; j < DIMENSION; j++) 
		{
			matrix[i * DIMENSION + j] = 0;
		}
	}
}

//cuda kernel for multiplying two matrices without tiling
__global__ void matrix_mul_kernel(int* a, int* b, int* c)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	
	//check if thread directly maps to the dimensions of resulting matrix
	if (row < DIMENSION && col < DIMENSION)
	{
		int result = 0;
		int k;
		for (k = 0; k < DIMENSION; k++)
		{
			result += (a[row * DIMENSION + k] * b[k * DIMENSION + col]);
		}
		c[row * DIMENSION + col] = result;
	}
}


int main(int argc, char **argv)
{
    int size = DIMENSION * DIMENSION;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

	//declare host and device matrices pointers
	int* mat_a;
	int* mat_b;
	int* mat_c;
	int* d_mat_a;
	int* d_mat_b;
	int* d_mat_c;
	
	//allocate memory for host matrices
	mat_a = (int*)malloc(size * sizeof(int));
	mat_b = (int*)malloc(size * sizeof(int));
	mat_c = (int*)malloc(size * sizeof(int));
	
	int i, j;
	
	initializeMatrix(mat_a);
	initializeMatrix(mat_b);
    clearMatrix(mat_c);

	//allocate matrices memeory on device
	hipMalloc((void **)&d_mat_a, size * sizeof(int));
	hipMalloc((void **)&d_mat_b, size * sizeof(int));
	hipMalloc((void **)&d_mat_c, size * sizeof(int));

	//copy A and B matrices from host to device
	hipMemcpy(d_mat_a, mat_a, size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_mat_b, mat_b, size * sizeof(int), hipMemcpyHostToDevice);

	//execute cuda kernel
    dim3 threadsPerBlock(DIMENSION, DIMENSION);
    dim3 blocksPerGrid(1, 1);
        if (size > 512){
            threadsPerBlock.x = 512;
            threadsPerBlock.y = 512;
            blocksPerGrid.x = ceil(double(DIMENSION)/double(threadsPerBlock.x));
            blocksPerGrid.y = ceil(double(DIMENSION)/double(threadsPerBlock.y));
        }

    hipEventRecord(start);
    matrix_mul_kernel<<<blocksPerGrid,threadsPerBlock>>>(d_mat_a, d_mat_b, d_mat_c);
    hipEventRecord(stop);

	//copy the compute matrix C from device to host
	hipMemcpy(mat_c, d_mat_c, size * sizeof(int), hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
	
	//free cuda memory
	hipFree(d_mat_a);
	hipFree(d_mat_b);
	hipFree(d_mat_c);

	//print matrix A
    printf("Matrix A: \n");
	for (i = 0; i < DIMENSION; i++)
	{
		for (j = 0; j < DIMENSION; j++)
		{
			printf("%d ", mat_a[i * DIMENSION + j]);
		}
		printf("\n");
	}
    printf("\n");
    //print matrix B
    printf("Matrix B: \n");
	for (i = 0; i < DIMENSION; i++)
	{
		for (j = 0; j < DIMENSION; j++)
		{
			printf("%d ", mat_b[i * DIMENSION + j]);
		}
		printf("\n");
	}
    printf("\n");
    //print the resulting matrix
    printf("Matrix C: \n");
	for (i = 0; i < DIMENSION; i++)
	{
		for (j = 0; j < DIMENSION; j++)
		{
			printf("%d ", mat_c[i * DIMENSION + j]);
		}
		printf("\n");
	}
    printf("\n");
    printf("Time [ms]: %f \n", milliseconds);
}

